#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "./metrictime.hpp"

#define block_dim 256

__global__ void mean_array(int *d_memory, double *d_resultados, int k, int m)
{ 
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id > m * 4) return;

    int array_id = thread_id / 4;
    int op_id = thread_id % 4;

    double resultado = 0;
    if (op_id == 0){
        for (int i = 0; i < k; i++){
            resultado += d_memory[array_id * k + i];
        }
        resultado = resultado / k;
    }
    else if (op_id == 1){
        resultado = d_memory[array_id * k];
         for (int i = 1; i < k; i++){
            if (d_memory[array_id * k + i] > resultado)
                resultado = d_memory[array_id * k + i];
        }
    }
    else if (op_id == 2){
        resultado = d_memory[array_id * k];
        for (int i = 1; i < k; i++){
            if (resultado > d_memory[array_id * k + i])
                resultado = d_memory[array_id * k + i];
        }
    }
    if (op_id == 3){
        double prom = 0;
        for (int i = 0; i < k; i++){
            prom += d_memory[array_id * k + i];
        }
        prom = prom / k;
        for (int i = 0; i < k; i++){
            float aux = d_memory[array_id * k + i] - prom;
            resultado += aux * aux;
        }
        resultado = resultado / k;
        resultado = sqrt(resultado);
        
    }
    d_resultados[array_id * 4 + op_id] = resultado;
}

int main(int argc, char const *argv[])
{
    if(argc != 3 ) {
        std::cout << "🤨🤨🤨" << std::endl;
        return 1; 
    }
    unsigned int m = atoi(argv[1]);
    unsigned int k = 2 << atoi(argv[2]);

    long long tamano = m * k;
    int * h_memory = (int *) malloc(m * k * sizeof(int));                // array del host
    double * h_resultados = (double *) malloc(m * 4 * sizeof(double));   // aquí se guardan los resultados
    int *d_memory;                                                      // array de la gpu, se copian el array del host
    double *d_resultados;                                               // aquí se guardan los resultados

    for (int i = 0; i < m; i++)
    {
        for (long long int j = 0; j < k; j++)
        {
            h_memory[i * k + j] = random()%(k + 1);
        }
    }

    hipMalloc((void **)&d_memory, tamano * sizeof(int));       // robando memoria 🥷  🤑
    hipMalloc((void **)&d_resultados, m * 4 * sizeof(double)); // robando memoria 🥷

    TIMERSTART(CUDA);

    hipMemcpy(d_memory, h_memory, tamano * sizeof(int), hipMemcpyHostToDevice);

    dim3 blkDim(block_dim, 1, 1);
    dim3 grdDim((m * 4 + block_dim - 1)/block_dim, 1, 1);
    mean_array<<<grdDim, blkDim>>>(d_memory, d_resultados, k, m);
    
    hipDeviceSynchronize();

    hipMemcpy(h_resultados, d_resultados, m * 4 * sizeof(double), hipMemcpyDeviceToHost);

    TIMERSTOP(CUDA);
    /*
    for (long long i = 0; i < m; i++)
    {
        std::cout << "Mean: " << h_resultados[i * 4];
        std::cout << ", Max: " << h_resultados[i * 4 + 1];
        std::cout << ", Min: " << h_resultados[i * 4 + 2];
        std::cout << ", Desv: " << h_resultados[i * 4 + 3] << std::endl;
    }
    */
    hipFree(d_resultados);
    hipFree(d_memory);
    free(h_memory);
    free(h_resultados);
    return 0;
}