#include <hip/hip_runtime.h>
#include <iostream>
// vo eri bueno🤨🤨🤨🤨🤨🤨🤨
// asi era la wea o no ? con .cu? .culia

// static int block_dim = 128; // hebras por

__global__ void mean_array(int *d_memory, double *d_resultados, int k)
{ // esta wea es cudaaaaaaaaaaaa se  pera 1 seg 😶‍🌫️😶‍🌫️😶‍🌫️😶‍🌫️😶‍🌫️😶‍🌫️😶‍🌫️😶‍🌫️
    int id_array = blockIdx.x;
    int id_thread = threadIdx.x;

    double resultado = 0;
    if (id_thread == 0)
    {
        for (int i = 0; i < k; i++)
        {
            resultado += d_memory[id_array * k + i];
        }
        resultado = resultado / k;
    }
    else if (id_thread == 1)
    {
    }
    else if (id_thread == 2)
    {
    }
    else if (id_thread == 3)
    {
    }
    switch (id_thread)
    {
    case (0):
        for (int i = 0; i < k; i++)
        {
            resultado += d_memory[id_array * k + i];
        }
        resultado = resultado / k;
        break;
    case (1):
        resultado = d_memory[id_array * k];
        for (int i = 1; i < k; i++)
        {
            if (d_memory[id_array * k + i] > resultado)
                resultado = d_memory[id_array * k + i];
        }
        break;
    case (2):
        resultado = d_memory[id_array * k];
        for (int i = 1; i < k; i++)
        {
            if (resultado > d_memory[id_array * k + i])
                resultado = d_memory[id_array * k + i];
        }
        break;
    case (3):
        for (int i = 0; i < k; i++)
        {
            float aux = d_memory[id_array * k + i] - d_resultados[id_array * 4];
            resultado += aux * aux;
        }
        resultado = resultado / k;
        resultado = sqrt(resultado);
        break;
    }
    d_resultados[id_array * 4 + id_thread] = resultado;
}

int main()
{
    int m = 10;
    int k = 10000;

    int tamano = m * k;
    int h_memory[m * k];        // array del host
    double h_resultados[m * 4]; // aquí se guardan los resultados
    int *d_memory;              // array de la gpu, se copian el array del host
    double *d_resultados;       // aquí se guardan los resultados

    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < k; j++)
        {
            h_memory[i * k + j] = j;
        }
    }

    hipMalloc((void **)&d_memory, tamano * sizeof(int));       // robando memoria 🥷
    hipMalloc((void **)&d_resultados, m * 4 * sizeof(double)); // robando memoria 🥷

    hipMemcpy(d_memory, h_memory, tamano * sizeof(int), hipMemcpyHostToDevice);

    dim3 blkDim(4, 1, 1);
    dim3 grdDim(m, 1, 1);

    mean_array<<<grdDim, blkDim>>>(d_memory, d_resultados, k);

    hipMemcpy(h_resultados, d_resultados, m * 4 * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < m; i++)
    {
        std::cout << "Mean: " << h_resultados[i * 4];
        std::cout << ", Max: " << h_resultados[i * 4 + 1];
        std::cout << ", Min: " << h_resultados[i * 4 + 2];
        std::cout << ", Desv: " << h_resultados[i * 4 + 3] << std::endl;
    }

    hipFree(&d_resultados);
    hipFree(&d_memory);

    return 0;
}